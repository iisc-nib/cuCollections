#include "hip/hip_runtime.h"
/*
-- TPC-H Query 9

select
        nation,
        o_year,
        sum(amount) as sum_profit
from
        (
                select
                        n_name as nation,
                        extract(year from o_orderdate) as o_year,
                        l_extendedprice * (1 - l_discount) - ps_supplycost * l_quantity as amount
                from
                        part,
                        supplier,
                        lineitem,
                        partsupp,
                        orders,
                        nation
                where
                        s_suppkey = l_suppkey
                        and ps_suppkey = l_suppkey
                        and ps_partkey = l_partkey
                        and p_partkey = l_partkey
                        and o_orderkey = l_orderkey
                        and s_nationkey = n_nationkey
                        and p_name like '%green%'
        ) as profit
group by
        nation,
        o_year
order by
        nation,
        o_year desc

*/

#include "utils.h"

#include <cuco/static_map.cuh>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/logical.h>
#include <thrust/sequence.h>
#include <thrust/tuple.h>

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>

#include <arrow/array.h>
#include <arrow/io/api.h>
#include <arrow/record_batch.h>
#include <arrow/table.h>
#include <parquet/arrow/reader.h>

#include <iomanip>
#include <iostream>

CUCO_DECLARE_BITWISE_COMPARABLE(double);

#define TILE_SIZE 1
namespace cg = cooperative_groups;
template <typename Map>
__global__ void build_hash_primary_key(Map map_ref, int32_t* nationkey, size_t nationsize)
{
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= nationsize) return;
  auto this_thread = cg::tiled_partition<TILE_SIZE>(cg::this_thread_block());
  map_ref.insert(this_thread, cuco::pair{nationkey[tid], tid});
}

struct multijoin_t {
  int32_t n_idx;
  int32_t s_idx;
  int32_t ps_idx;
};

template <typename ProbeMap>
__global__ void probe_partsupp_size(ProbeMap s_map_ref,
                                    ProbeMap n_map_ref,
                                    int32_t* ps_suppkey,
                                    int32_t* s_suppkey,
                                    int32_t* s_nationkey,
                                    int32_t* n_nationkey,
                                    int32_t* res_size,
                                    size_t ps_size)
{
  int32_t tid = (threadIdx.x + blockIdx.x * blockDim.x);
  if (tid >= ps_size) return;
  // now join the part supp with supplier
  auto s_idx = s_map_ref.find(ps_suppkey[tid]);
  if (s_idx == s_map_ref.end()) return;
  // get the nation idx, based on the supplier idx
  auto n_idx = n_map_ref.find(s_nationkey[s_idx->second]);
  if (n_idx == n_map_ref.end()) return;
  // append n,s,ps indices to the result
  atomicAdd(res_size, 1);
}

template <typename ProbeMap>
__global__ void probe_partsupp(ProbeMap s_map_ref,
                               ProbeMap n_map_ref,
                               int32_t* ps_suppkey,
                               int32_t* s_suppkey,
                               int32_t* s_nationkey,
                               int32_t* n_nationkey,
                               multijoin_t* result,
                               int32_t* res_idx,
                               size_t ps_size)
{
  int32_t tid = (threadIdx.x + blockIdx.x * blockDim.x);
  if (tid >= ps_size) return;
  // now join the part supp with supplier
  auto s_idx = s_map_ref.find(ps_suppkey[tid]);
  if (s_idx == s_map_ref.end()) return;
  // get the nation idx, based on the supplier idx
  auto n_idx = n_map_ref.find(s_nationkey[s_idx->second]);
  if (n_idx == n_map_ref.end()) return;
  // append n,s,ps indices to the result
  auto idx           = atomicAdd(res_idx, 1);
  result[idx].n_idx  = n_idx->second;
  result[idx].s_idx  = s_idx->second;
  result[idx].ps_idx = tid;
}

int main(int argc, const char** argv)
{
  std::cout << std::setprecision(10);

  std::string dbDir         = getDataDir(argv, argc);
  std::string lineitem_file = dbDir + "lineitem.parquet";
  std::string part_file     = dbDir + "part.parquet";
  std::string orders_file   = dbDir + "orders.parquet";
  std::string nation_file   = dbDir + "nation.parquet";
  std::string supplier_file = dbDir + "supplier.parquet";
  std::string partsupp_file = dbDir + "partsupp.parquet";

  auto lineitem_table = getArrowTable(lineitem_file);
  auto part_table     = getArrowTable(part_file);
  auto orders_table   = getArrowTable(orders_file);
  auto nation_table   = getArrowTable(nation_file);
  auto supplier_table = getArrowTable(supplier_file);
  auto partsupp_table = getArrowTable(partsupp_file);

  size_t lineitem_size = lineitem_table->num_rows();
  size_t part_size     = part_table->num_rows();
  size_t orders_size   = orders_table->num_rows();
  size_t nation_size   = nation_table->num_rows();
  size_t supplier_size = supplier_table->num_rows();
  size_t partsupp_size = partsupp_table->num_rows();

  // now we need to semi-materialize the join
  // first join supplier and nation and semi materialize into a separate table.
  auto n_nationkey_map =
    cuco::static_map{nation_size * 2,
                     cuco::empty_key{(int32_t)-1},
                     cuco::empty_value{(int32_t)-1},
                     thrust::equal_to<int32_t>{},
                     cuco::linear_probing<TILE_SIZE, cuco::default_hash_function<int32_t>>()};

  auto s_supplierkey_map =
    cuco::static_map{supplier_size * 2,
                     cuco::empty_key{(int32_t)-1},
                     cuco::empty_value{(int32_t)-1},
                     thrust::equal_to<int32_t>{},
                     cuco::linear_probing<TILE_SIZE, cuco::default_hash_function<int32_t>>()};
  int TB = 1024;
  int32_t *n_nationkey, *d_n_nationkey;
  int32_t *s_supplierkey, *d_s_supplierkey;
  int32_t *s_nationkey, *d_s_nationkey;
  int32_t *ps_suppkey, *d_ps_suppkey;
  n_nationkey   = read_column_typecasted<int32_t>(nation_table, "n_nationkey");
  s_supplierkey = read_column_typecasted<int32_t>(supplier_table, "s_suppkey");
  s_nationkey   = read_column_typecasted<int32_t>(supplier_table, "s_nationkey");
  ps_suppkey    = read_column_typecasted<int32_t>(partsupp_table, "ps_suppkey");

  hipMalloc(&d_n_nationkey, nation_size * sizeof(int32_t));
  hipMalloc(&d_s_supplierkey, supplier_size * sizeof(int32_t));
  hipMalloc(&d_s_nationkey, supplier_size * sizeof(int32_t));
  hipMalloc(&d_ps_suppkey, partsupp_size * sizeof(int32_t));
  hipMemcpy(d_n_nationkey, n_nationkey, nation_size * sizeof(int32_t), hipMemcpyHostToDevice);
  hipMemcpy(
    d_s_supplierkey, s_supplierkey, supplier_size * sizeof(int32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_s_nationkey, s_nationkey, supplier_size * sizeof(int32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_ps_suppkey, ps_suppkey, partsupp_size * sizeof(int32_t), hipMemcpyHostToDevice);

  build_hash_primary_key<<<getGridSize(nation_size, TB), TB>>>(
    n_nationkey_map.ref(cuco::insert), d_n_nationkey, nation_size);

  build_hash_primary_key<<<getGridSize(supplier_size, TB), TB>>>(
    s_supplierkey_map.ref(cuco::insert), d_s_supplierkey, supplier_size);
  hipDeviceSynchronize();
  int32_t join_size, *d_join_size;
  hipMalloc(&d_join_size, sizeof(int32_t));
  hipMemset(d_join_size, 0, sizeof(int32_t));
  probe_partsupp_size<<<getGridSize(partsupp_size, TB), TB>>>(s_supplierkey_map.ref(cuco::find),
                                                              n_nationkey_map.ref(cuco::find),
                                                              d_ps_suppkey,
                                                              d_s_supplierkey,
                                                              d_s_nationkey,
                                                              d_n_nationkey,
                                                              d_join_size,
                                                              partsupp_size);
  hipMemcpy(&join_size, d_join_size, sizeof(int32_t), hipMemcpyDeviceToHost);
  multijoin_t *n_s_ps_join, *d_n_s_ps_join;
  hipMalloc(&d_n_s_ps_join, sizeof(multijoin_t)*join_size);
  hipMemset(d_join_size, 0, sizeof(int32_t));
  probe_partsupp<<<getGridSize(partsupp_size, TB), TB>>>(s_supplierkey_map.ref(cuco::find),
                                                              n_nationkey_map.ref(cuco::find),
                                                              d_ps_suppkey,
                                                              d_s_supplierkey,
                                                              d_s_nationkey,
                                                              d_n_nationkey,
                                                              d_n_s_ps_join,
                                                              d_join_size,
                                                              partsupp_size);
  n_s_ps_join = (multijoin_t*)malloc(sizeof(multijoin_t)*join_size);
  hipMemcpy(n_s_ps_join, d_n_s_ps_join, sizeof(multijoin_t)*join_size, hipMemcpyDeviceToHost);

  for (size_t i=0; i<join_size; i++) {
    auto n_idx = n_s_ps_join[i].n_idx;
    auto s_idx = n_s_ps_join[i].s_idx;
    auto ps_idx = n_s_ps_join[i].ps_idx;
    std::cout << n_nationkey[n_idx] << " " << s_nationkey[s_idx] << " " << s_supplierkey[s_idx] << " " << 
      ps_suppkey[ps_idx] << "\n";
  }

  std::cout << "nationkey size: " << n_nationkey_map.size() << "\n";
  std::cout << "supplierkey size: " << s_supplierkey_map.size() << "\n";
  std::cout << "nation, supplier, partsupplier sizes: " << nation_size << " " << supplier_size << " " << partsupp_size << "\n";
  std::cout << "supplier, nation and partsupplier join size: " << join_size << "\n";
}